#include "hip/hip_runtime.h"
//opencv_cuda.cu:ʹ���Զ��庯����ʵ��cuda�汾ͼƬ��ת
//authored by alpc40,Bizat
//version��visual studio 2019\cuda toolkit 11.0\openCV 4.4.0
#include "opencv2/opencv.hpp"
#include "hip/hip_runtime.h"
#include ""
#include <stdio.h>

#include<iostream>
using namespace std;
using namespace cv;
#ifdef _DEBUG
#pragma comment ( lib,"opencv_core440d.lib")
#pragma comment ( lib,"opencv_highgui440d.lib")
#pragma comment ( lib,"opencv_calib3d440d.lib")
#pragma comment ( lib,"opencv_imgcodecs440d.lib")
#pragma comment ( lib,"opencv_imgproc440d.lib")
#pragma comment ( lib,"opencv_cudaimgproc440d.lib")
#pragma comment ( lib,"opencv_cudaarithm440d.lib")
#pragma comment ( lib,"cudart.lib")
#else
#pragma comment ( lib,"opencv_core320.lib")
#pragma comment ( lib,"opencv_highgui320.lib")
#pragma comment ( lib,"opencv_calib3d320.lib")
#pragma comment ( lib,"opencv_imgcodecs320.lib")
#pragma comment ( lib,"opencv_imgproc320.lib")
#pragma comment ( lib,"opencv_cudaimgproc320.lib")
#pragma comment ( lib,"opencv_cudaarithm320.lib")
#pragma comment ( lib,"cudart.lib")
#endif
//��������
#define CHECK_ERROR(call){\
    const hipError_t err = call;\
    if (err != hipSuccess)\
    {\
        printf("Error:%s,%d,",__FILE__,__LINE__);\
        printf("code:%d,reason:%s\n",err,hipGetErrorString(err));\
        exit(1);\
    }\
}
//�ں˺�����ʵ�����·�ת
__global__ void swap_image_kernel(cuda::PtrStepSz<uchar3> cu_src, cuda::PtrStepSz<uchar3> cu_dst, int h, int w)
{
    //����ķ������ο�ǰ������
    unsigned int x = blockDim.x * blockIdx.x + threadIdx.x;
    unsigned int y = blockDim.y * blockIdx.y + threadIdx.y;
    //ΪɶҪ�������ƣ��ο�ǰ������
    if (x < cu_src.cols && y < cu_src.rows)
    {
        //Ϊ�β���h-y-1,������h-y���Լ�˼��Ŷ
        cu_dst(y, x) = cu_src(h - y - 1, x);
    }
}
//���ú�������Ҫ����block��grid�Ĺ�ϵ
void swap_image(cuda::GpuMat src, cuda::GpuMat dst, int h, int w)
{
    assert(src.cols == w && src.rows == h);
    int uint = 32;
    //�ο�ǰ�����ĵ�block��grid�ļ��㷽����ע�ⲻҪ����GPU����
    dim3 block(uint, uint);
    dim3 grid((w + block.x - 1) / block.x, (h + block.y - 1) / block.y);
    printf("grid = %4d %4d %4d\n", grid.x, grid.y, grid.z);
    printf("block= %4d %4d %4d\n", block.x, block.y, block.z);
    swap_image_kernel << <grid, block >> > (src, dst, h, w);
    //ͬ��һ�£���Ϊ���������ܴܺ�
    CHECK_ERROR(hipDeviceSynchronize());
}
int main(int argc, char** argv)
{
    Mat src, dst;
    cuda::GpuMat cu_src, cu_dst;
    int h, w;
    //����argv[1]����ͼƬ���ݣ�BGR��ʽ������
    src = imread("C:\\Users\\Bizat\\Pictures\\��Ѿͷ.jpg");
    //����Ƿ���ȷ����
    if (src.data == NULL)
    {
        cout << "Read image error" << endl;
        return -1;
    }
    h = src.rows; w = src.cols;
    cout << "ͼƬ�ߣ�" << h << ",ͼƬ��" << w << endl;
    //�ϴ�CPUͼ�����ݵ�GPU����hipMalloc��hipMemcpy����Ŷ����ʵupload���������ôд��
    cu_src.upload(src);
    //����GPU�ռ䣬Ҳ���Ե����������룬����������Ҫ���룬Ҫ��Ȼ�ں˺����ᱬ��Ŷ
    cu_dst = cuda::GpuMat(h, w, CV_8UC3, Scalar(0, 0, 0));
    //����CPU�ռ�
    dst = Mat(h, w, CV_8UC3, Scalar(0, 0, 0));
    //���ú���swap_image,�ɸú��������ں˺�����������η����������׳���
    //��Ȼ��Ҳ����ֱ������������ں˺���������̫�����������
    swap_image(cu_src, cu_dst, h, w);
    //����GPU���ݵ�CPU����upload()��Ӧ
    cu_dst.download(dst);
    //��ʾcpuͼ�������װ��openCV������openGL,�ǿ���ֱ����ʾGpuMat
    imshow("dst", dst);
    //�ȴ�����
    waitKey();
    //дͼƬ���ļ�
    if (argc == 3)
        imwrite("C:\\Users\\Bizat\\Pictures\\��Ѿͷ2.jpg", dst);
    return 0;
}